#include "common.h"
#include "costs.h"
#include "util.h"


static PixType *d_imgleft_data, *d_imgright_data;
static PixType *d_imgleft_grad, *d_imgright_grad; 
static PixType *d_clibTab;
static CostType *d_pixDiff;
static CostType *d_cost;
static CostType *d_hsum;


static int p1, p2;
static int blocksize;
static int rows, cols, img_size;
static int preFilterCap;
const int TAB_OFS = 256*4, TAB_SIZE = 256 + TAB_OFS*2; 

void cuda_init(SGM_PARAMS *params)
{
	preFilterCap = params->preFilterCap; 
	p1 = params->P1;
	p2 = params->P2;
	blocksize = params->BlockSize;
}

cv::Mat compute_disparity(cv::Mat *left_img, cv::Mat *right_img, float *cost_time)
{
	if(CV_8UC1 !=  left_img->type() || CV_8UC1 != right_img->type())
	{
		std::cout<<"image type error\n";
		exit(-1);
	}
	static bool is_first_called = true;
	if(is_first_called)
	{
		std::cout<<"First Called\n";
		is_first_called = false;
		rows = left_img->rows;
		cols = left_img->cols;
		img_size = rows * cols;
		CUDA_CHECK_RETURN(hipMalloc((void **)&d_imgleft_data, sizeof(PixType) * img_size));	
		CUDA_CHECK_RETURN(hipMalloc((void **)&d_imgright_data, sizeof(PixType) * img_size));	
		CUDA_CHECK_RETURN(hipMalloc((void **)&d_imgleft_grad, sizeof(PixType) * img_size));	
		CUDA_CHECK_RETURN(hipMalloc((void **)&d_imgright_grad, sizeof(PixType) * img_size));	
		CUDA_CHECK_RETURN(hipMalloc((void **)&d_clibTab, sizeof(PixType) * TAB_SIZE));	
		CUDA_CHECK_RETURN(hipMalloc((void **)&d_pixDiff, sizeof(CostType) * img_size * MAX_DISPARITY));	
		CUDA_CHECK_RETURN(hipMalloc((void **)&d_cost, sizeof(CostType) * img_size * MAX_DISPARITY));	
		CUDA_CHECK_RETURN(hipMalloc((void **)&d_hsum, sizeof(CostType) * img_size * MAX_DISPARITY));	

		fill_tab<<<1, 1>>>(d_clibTab, TAB_SIZE, TAB_OFS, preFilterCap);
	}
	
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start, 0);
	CUDA_CHECK_RETURN(hipMemcpyAsync(d_imgleft_data, left_img->ptr<PixType>(), sizeof(PixType) * img_size, hipMemcpyHostToDevice));
	CUDA_CHECK_RETURN(hipMemcpyAsync(d_imgright_data, right_img->ptr<PixType>(), sizeof(PixType) * img_size, hipMemcpyHostToDevice));
	
	get_gradient<<<rows, WARP_SIZE>>>(d_imgleft_data, d_imgright_data, d_imgleft_grad, d_imgright_grad, d_clibTab + TAB_OFS, rows, cols);


	CUDA_CHECK_RETURN(hipMemset(d_pixDiff, 0, sizeof(CostType) * img_size * MAX_DISPARITY));
	get_pixel_diff<<<rows, MAX_DISPARITY>>>(d_imgleft_grad, d_imgright_grad, rows, cols, 0, d_pixDiff); 
	get_pixel_diff<<<rows, MAX_DISPARITY>>>(d_imgleft_data, d_imgright_data, rows, cols, 2, d_pixDiff); 

	get_hsum<<<rows, MAX_DISPARITY>>>(d_pixDiff, d_hsum, rows, cols, blocksize);
	get_cost<<<cols - MAX_DISPARITY, MAX_DISPARITY>>>(d_hsum, d_cost, p2, rows, cols, blocksize);//d_cost前MAX_DISPARITY列没有用

	hipEventRecord(stop, 0);
	hipDeviceSynchronize();
	float cost_t = 0;
	hipEventElapsedTime(&cost_t, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	printf("alogrithm cost:%fms\n", cost_t);
#if 0
	
	double cpy_start = cv::getTickCount();
	PixType *h_grad = (PixType *)malloc(sizeof(PixType) * img_size);
	if(!h_grad)
	{
		printf("error\n");
	}
	CUDA_CHECK_RETURN(hipMemcpy(h_grad, d_imgleft_grad, sizeof(PixType) * img_size, hipMemcpyDeviceToHost));
	double cpy_end = cv::getTickCount();
	printf("copy data cost:%lfms\n", (cpy_end - cpy_start)*1000/cv::getTickFrequency());

	ofstream  gradient_file;
	gradient_file.open("grad.out", ios::out);
	for(int i= 0 ; i < rows; i++)
	for(int j = 0; j < cols; j++ )
			gradient_file<<"grad[row="<<i<<" col="<<j<<"]="<<(int)h_grad[i * cols +j]<<endl;
	gradient_file.close();
	free(h_grad);
#endif

#if 1
	
	double cpy_start = cv::getTickCount();
	CostType *h_cost = (CostType *)malloc(sizeof(CostType) * img_size * MAX_DISPARITY);
	if(!h_cost)
	{
		printf("error\n");
	}
	CUDA_CHECK_RETURN(hipMemcpy(h_cost, d_cost, sizeof(CostType) * img_size * MAX_DISPARITY, hipMemcpyDeviceToHost));
	double cpy_end = cv::getTickCount();
	printf("copy data cost:%lfms\n", (cpy_end - cpy_start)*1000/cv::getTickFrequency());


	ofstream  cost0;
	cost0.open("cost.out", ios::out);
	for(int i=0;i<rows;i++)
	for(int j = MAX_DISPARITY; j < cols; j++ )
		for(int k=0; k < MAX_DISPARITY; k++)
			cost0<<"cost[row="<<i<<" col="<<j<<" d="<<k<<"]: "<<h_cost[(i * cols + j)*MAX_DISPARITY + k]<<endl;
	cost0.close();
	free(h_cost);
#endif

	
	return *left_img;
}


void free_gpu_mem()
{
	std::cout<<"Free Mem\n";
	CUDA_CHECK_RETURN(hipFree(d_imgleft_data));
	CUDA_CHECK_RETURN(hipFree(d_imgright_data));
	CUDA_CHECK_RETURN(hipFree(d_clibTab));
}
